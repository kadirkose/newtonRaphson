#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <termios.h>
#include <fcntl.h>
#include <unistd.h>
#include <string.h>
#include <time.h>
#include <gsl/gsl_poly.h>
#include <gsl/gsl_complex.h>

#define floatSize 32
#define maxFloatNumber 32
#define CPLD_CLOCK_FREQ 125000000

int degree_min, degree_max;
int cpldUart;
__uint32_t iteration_count;

typedef union {
    float f;
    struct {
        // Order is important.
        // Here the members of the union data structure
        // use the same memory (32 bits).
        // The ordering is taken
        // from the LSB to the MSB.
        unsigned int mantissa : 23;
        unsigned int exponent : 8;
        unsigned int sign : 1;
    } raw;
	__uint32_t bit_vector;
} floatingPoint;

floatingPoint fPoint[maxFloatNumber];
floatingPoint fPointTemp;
floatingPoint errRate;

void printBinary(int n, int i)
{
    // Prints the binary representation
    // of a number n up to i-bits.
    int k;
    for (k= i-1; k >= 0; k--) {
        if((n >> k) & 1)
            printf("1");
        else
            printf("0");
    }
}

int sendToCpld()
{
	int degreeCount;
	char degree_min_pos;
	char deg_max;

	if(degree_min < 0)
		degree_min_pos= -1*degree_min;
	else
		degree_min_pos= 0;

	deg_max = degree_min_pos + degree_max;

	write(cpldUart, &deg_max, 1);
	write(cpldUart, &fPointTemp.bit_vector, 4);
	write(cpldUart, &errRate.bit_vector, 4);
	write(cpldUart, &iteration_count, 4);

	for(degreeCount= 0; degreeCount <= (degree_max-degree_min); degreeCount++)
	{
		write(cpldUart, &fPoint[degreeCount].bit_vector , 4);
	}

	return 0;
}

int receiveFromCpld()
{
	char rcvData[4];
	__uint32_t rcvDataInt;
	__uint32_t cycleCount;
	float fRcvData;
	double cpld_compute_time;

	if(read(cpldUart, rcvData, 4) > 0)
	{
		memcpy(&rcvDataInt, (unsigned int*)&rcvData, sizeof(rcvData));
		memcpy(&fRcvData,   &rcvData, sizeof(rcvData));
		printf("FPGA Result for given variable substitution :                 %.040f \n",fRcvData);
	}
	else
		return 1;

	if(read(cpldUart, rcvData, 4) > 0)
	{
		memcpy(&cycleCount, (unsigned int*)&rcvData, sizeof(rcvData));
		printf("FPGA operation cycle count :                                  ");
		printf("%u\n", cycleCount);
		cpld_compute_time = (double)cycleCount / 125;
		printf("Computation time of fpga ignoring the uart communication:     ");
		printf("%lf usec\n",cpld_compute_time);
	}
	else
		return 1;

	if(read(cpldUart, rcvData, 4) > 0)
	{
		memcpy(&rcvDataInt, (unsigned int*)&rcvData, sizeof(rcvData));
		printf("FPGA computation iteration count      :                       ");
		printf("%u\n", rcvDataInt);
	}
	else
		return 1;

	if(read(cpldUart, rcvData, 4) > 0)
	{
		memcpy(&rcvDataInt, (unsigned int*)&rcvData, sizeof(rcvData));
		printf("FPGA total cycle count      :                                 ");
		printf("%u\n", rcvDataInt);
	}
	else
		return 1;

	return 0;
}

int computeDerivative()
{
	int degreeCount;
	float subsResult=0, derSubsResult=0, err=0, variable;
	int iter_count = 0;
	struct timespec ts_start_compute;
	struct timespec ts_stop_compute;
	double cpu_compute_time, cpu_power_compute_time, cpu_mult_compute_time, cpu_variable_compute_time;
	int i;
	float power_result[33];
	struct timespec ts_start_power_compute, ts_stop_power_compute;
	struct timespec ts_start_mult_compute, ts_stop_mult_compute;
	struct timespec ts_start_variable_compute, ts_stop_variable_compute;
	
	variable = fPointTemp.f;

	double dPoly[degree_max+1];
	double roots[degree_max*2];
	float fPoly[degree_max+1];

	// memcpy(fPoly, fPoint, sizeof(float)*(degree_max+1));

	// for (i = 0; i < degree_max+1; i++)
	// 	dPoly[i] = fPoly[i];
	
  	// gsl_poly_complex_workspace *w  = gsl_poly_complex_workspace_alloc (degree_max+1);
  
  	// clock_gettime(CLOCK_MONOTONIC, &ts_start_compute);
  	// gsl_poly_complex_solve (dPoly, degree_max+1, w, roots);
	// clock_gettime(CLOCK_MONOTONIC, &ts_stop_compute);

  	// gsl_poly_complex_workspace_free (w);

	// cpu_compute_time = (ts_stop_compute.tv_sec - ts_start_compute.tv_sec)*1000000 + (ts_stop_compute.tv_nsec - ts_start_compute.tv_nsec) / 1000;
	// printf("Time spent for computation CPU using gsl                 : %lf usec \n\n\n",cpu_compute_time);

  	// for (i = 0; i < degree_max; i++)
    //   printf("%d.root                                   		     : %+.18f %+.18fi\n", i, roots[2*i], roots[2*i+1]);


	clock_gettime(CLOCK_MONOTONIC, &ts_start_compute);
	
	while(iter_count < iteration_count)
	{
		subsResult = 0;
		derSubsResult = 0;
		clock_gettime(CLOCK_MONOTONIC, &ts_start_power_compute);
		for(degreeCount= 0; degreeCount < (degree_max - degree_min + 2); degreeCount++)
		{
			power_result[degreeCount] = pow(variable, (degreeCount + degree_min - 1));
		
		}
		clock_gettime(CLOCK_MONOTONIC, &ts_stop_power_compute);
		for(degreeCount= 0; degreeCount < (degree_max - degree_min + 1); degreeCount++)
		{
			if((variable == 0) && (degree_min != 0))
			{
				printf("Process is stopped because variable is zero. x/0 is nan");
				goto exit;
			}
			subsResult+= (fPoint[degreeCount].f) * power_result[degreeCount + 1];
			if(degreeCount + degree_min == 0)
				derSubsResult+= 0;
			else
				derSubsResult+= (fPoint[degreeCount].f * (degreeCount + degree_min)) * power_result[degreeCount];
		}
		clock_gettime(CLOCK_MONOTONIC, &ts_stop_mult_compute);
		if(derSubsResult == 0)
		{
			printf("Process is stopped because derSubsResult is zero. subsResult/0 is nan. iteratin: %d, root :%f \n", iter_count, variable);
			goto exit;
		}
		err = subsResult / derSubsResult;
		variable = variable - err;
		if(fabs(err) < errRate.f)
			break;
		iter_count += 1;
		clock_gettime(CLOCK_MONOTONIC, &ts_stop_variable_compute);
	}
exit:
	clock_gettime(CLOCK_MONOTONIC, &ts_stop_compute);
	printf(" ----------------------------- CPU ----------------------------------- \n");
	printf("Iteration Count : %d\n", iter_count);
	printf("Result in float : %.040f\n", variable);
	cpu_compute_time = (ts_stop_compute.tv_sec - ts_start_compute.tv_sec)*1000000000 + 
					   (ts_stop_compute.tv_nsec - ts_start_compute.tv_nsec) / 1000;
	cpu_power_compute_time = (ts_stop_power_compute.tv_sec - ts_start_power_compute.tv_sec)*1000000000 + 
							 (ts_stop_power_compute.tv_nsec - ts_start_power_compute.tv_nsec);
	cpu_mult_compute_time = (ts_stop_mult_compute.tv_sec - ts_stop_power_compute.tv_sec)*1000000000 + 
							(ts_stop_mult_compute.tv_nsec - ts_stop_power_compute.tv_nsec);
	cpu_variable_compute_time = (ts_stop_variable_compute.tv_sec - ts_stop_mult_compute.tv_sec)*1000000000 + 
					   (ts_stop_variable_compute.tv_nsec - ts_stop_mult_compute.tv_nsec);
	printf("Power Computition ---- Mult/Add Computition ---- New Root Computition ---- Total Spent -- \n");
	printf("   Time(nsec)              Time(nsec)                 Time(nsec)             Time(usec) -- \n");
	printf("   %.05f ",cpu_power_compute_time);
	printf("             %.05f ",cpu_mult_compute_time);
	printf("                 %.05f ",cpu_mult_compute_time);
	printf("             %.05f \n\n\n",cpu_compute_time);

	return 0;
}

__device__ void sumOfArray(float *array, int size)
{
	int index = threadIdx.x;

	if(index*2 + 1 < size)
		array[index] = __fadd_rn(array[index*2], array[index*2 + 1]);
	else
		array[index] = array[index*2];

	__syncthreads();
}


__device__ void evalutaionPolynomial(float *coeff_array, float variable, float *polynomial_mult)
{
	int index = threadIdx.x;
	float polynomial_pow ;
	
	if(index == 0)
		polynomial_mult[index] = coeff_array[index];
	else
	{
		if(variable < 0)
		{
			//variable = __fsub_rn(0.0f, variable);
			variable = -variable;
			polynomial_pow = __powf(variable, (float)index);
			if(index%2)
				polynomial_pow = -polynomial_pow;
				//polynomial_pow = __fsub_rn(0.0f, polynomial_pow);
		}
		else
			polynomial_pow = __powf(variable, (float)index);
	
		polynomial_mult[index] = __fmul_rn(coeff_array[index], polynomial_pow); 
	}
	__syncthreads();
}

__device__ void evalutaionDerivativeOfPolynomial(float *coeff_array, float variable, float *derivative_mult)
{
	int index = threadIdx.x;
	float derivative_pow;

	if(index == 0)
	{
		derivative_mult[index] = 0;
	}
	else if(index == 1)
	{
		derivative_mult[index] = coeff_array[index];
	}
	else
	{
		if(variable < 0)
		{
			//variable = __fsub_rn(0.0f, variable);
			variable = -variable;
			derivative_pow = __powf(variable, (float)(index - 1));
			if(!(index%2))
				derivative_pow = -derivative_pow;
				//derivative_pow = __fsub_rn(0.0f, derivative_pow);
		}
		else
			derivative_pow = __powf(variable, (float)(index - 1));
		
		derivative_mult[index] = __fmul_rn(coeff_array[index], index);
		derivative_mult[index] = __fmul_rn(derivative_mult[index], derivative_pow);
	}
	__syncthreads();

}

__global__ void newtonRaphsonIteration (float *coeff_array, float *variable, float *size,
										float *err, float *derivative_mult, float *polynomial_mult)
{
	int block_index = blockIdx.x;
	int index = threadIdx.x;
	int count = 0;
	float addition_iter = ceil(log2(*size));
	float size_of_adder;
	*err = 0;

	if(block_index)
		evalutaionPolynomial(coeff_array, *variable, polynomial_mult);
	else
		evalutaionDerivativeOfPolynomial(coeff_array, *variable, derivative_mult);

	size_of_adder = *size;
	while(count < addition_iter)
	{
		if(index < size_of_adder)
		{
			if(block_index)
				sumOfArray(polynomial_mult, size_of_adder);
			else
				sumOfArray(derivative_mult, size_of_adder);
		}
		count++;
		size_of_adder = ceil(size_of_adder/2);
	}
	if(block_index == 0 && index == 0)
	{
		*err = __fdiv_rn(polynomial_mult[0], derivative_mult[0]);
	}
}

int computeDerivativeGpu()
{
	float err=0, variable;
	int iter_count = 0;
	struct timespec ts_start_compute;
	struct timespec ts_stop_compute;
	double gpu_compute_time;
    int size = degree_max - degree_min + 1;
    float *coeff_array;
	float *size_gpu;
	float *variable_gpu;
	float addition_iter = ceil(log2(size));
	variable = fPointTemp.f;
	
	float fsize = size;
	float *err_gpu;
	float *derivative_mult;
    float *polynomial_mult;

    hipMalloc(&coeff_array, sizeof(float)*size);
	hipMalloc(&size_gpu, sizeof(float));
	hipMalloc(&variable_gpu, sizeof(float));
	hipMalloc(&err_gpu, sizeof(float));

	hipMalloc(&derivative_mult, sizeof(float)*size);
	hipMalloc(&polynomial_mult, sizeof(float)*size);

    hipMemcpy(size_gpu, &fsize, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(coeff_array, fPoint, size*sizeof(float), hipMemcpyHostToDevice);

    clock_gettime(CLOCK_MONOTONIC, &ts_start_compute);
	while(iter_count < iteration_count)
	{

		hipMemcpy(variable_gpu, &variable, sizeof(float), hipMemcpyHostToDevice);
		newtonRaphsonIteration<<<2,size>>>(coeff_array, variable_gpu, size_gpu, err_gpu, derivative_mult, polynomial_mult);
        hipDeviceSynchronize();
		hipMemcpy(&err, err_gpu, sizeof(float), hipMemcpyDeviceToHost);
		//printf("computeDerivativeGpu var %f err %f \n",variable, err);
		variable = variable - err;
		if(fabs(err) < errRate.f)
			break;
		iter_count += 1;
	}
	clock_gettime(CLOCK_MONOTONIC, &ts_stop_compute);

	hipFree(coeff_array);
	hipFree(size_gpu);
	hipFree(variable_gpu);
	hipFree(derivative_mult);
	hipFree(polynomial_mult);
	printf(" ----------------------------- GPU ----------------------------------- \n");
	printf("\nIteration Count : %d\n", iter_count);
	printf("Result in float : %.040f\n", variable);
	gpu_compute_time = (ts_stop_compute.tv_sec - ts_start_compute.tv_sec)*1000000 + (ts_stop_compute.tv_nsec - ts_start_compute.tv_nsec) / 1000;
	printf("Time spent for computation GPU:  %lf usec \n\n\n",gpu_compute_time);

	return 0;
}

int uartInit()
{
	cpldUart = open("/dev/ttyUSB0", O_RDWR);
	// Create new termios struct, we call it 'tty' for convention
	struct termios tty;

	// Read in existing settings, and handle any error
	if(tcgetattr(cpldUart, &tty) != 0) {
	  printf("Error from tcgetattr\n");
	  return 1;
	}

	tty.c_cflag &= ~PARENB; // Clear parity bit, disabling parity (most common)
	tty.c_cflag &= ~CSTOPB; // Clear stop field, only one stop bit used in communication (most common)
	tty.c_cflag &= ~CSIZE; // Clear all bits that set the data size
	tty.c_cflag |= CS8; // 8 bits per byte (most common)
	tty.c_cflag &= ~CRTSCTS; // Disable RTS/CTS hardware flow control (most common)
	tty.c_cflag |= CREAD | CLOCAL; // Turn on READ & ignore ctrl lines (CLOCAL = 1)

	tty.c_lflag &= ~ICANON;
	tty.c_lflag &= ~ECHO; // Disable echo
	tty.c_lflag &= ~ECHOE; // Disable erasure
	tty.c_lflag &= ~ECHONL; // Disable new-line echo
	tty.c_lflag &= ~ISIG; // Disable interpretation of INTR, QUIT and SUSP
	tty.c_iflag &= ~(IXON | IXOFF | IXANY); // Turn off s/w flow ctrl
	tty.c_iflag &= ~(IGNBRK|BRKINT|PARMRK|ISTRIP|INLCR|IGNCR|ICRNL); // Disable any special handling of received bytes

	tty.c_oflag &= ~OPOST; // Prevent special interpretation of output bytes (e.g. newline chars)
	tty.c_oflag &= ~ONLCR; // Prevent conversion of newline to carriage return/line feed

	tty.c_cc[VTIME] = 250;    // Wait for up to 1s (10 deciseconds), returning as soon as any data is received.
	tty.c_cc[VMIN] = 0;

	// Set in/out baud rate to be 115200
	cfsetispeed(&tty, 115200);
	cfsetospeed(&tty, 115200);

	// Save tty settings, also checking for error
	if (tcsetattr(cpldUart, TCSANOW, &tty) != 0) {
	  close(cpldUart);
	  printf("Error from tcsetattr\n");
	  return 1;
	}
	return 0;
}

int main()
{
	int error = 0;
	struct timespec ts_start_time;
	double total_fpga_computition_time;
	struct timespec ts_end_compute;
	int index = 0;
	float a = 5.0;
	degree_min = 0;

	for(index = 6; index < 32; index+=5)
	{
		printf("----------------------Polynom Degree: %d ---------------------------- \n",index - 1);
		fPointTemp.f = ((float)rand()/(float)(RAND_MAX)) * a;
		printf("Starting Point %f:\n", fPointTemp.f);

		printf("polynomial random coefficients : \n");
   		for (int i = 0; i < index; i++)
		{
			fPoint[i].f = ((float)rand()/(float)(RAND_MAX)) * a;
			printf("%f \n",fPoint[i].f);
		}
		printf("\n");
		degree_max = index - 1;

		iteration_count = 100000;
		errRate.f = 0.0;

		error = computeDerivative();
		if(error)
		{
			printf("ERROR computeDerivative! \n");
			return (1);
		}

		error = computeDerivativeGpu();
		if(error)
		{
			printf("ERROR computeDerivative! \n");
			return (1);
		}
		
		error = uartInit();
		if(error)
		{
			printf("ERROR uartInit!\n");
			printf("Check the ttyUSB number!\n");
			return (1);
		}

		error = sendToCpld();
		if(error)
		{
			printf("ERROR sendToCpld! \n");
			return (1);
		}
		printf(" ----------------------------- FPGA ----------------------------------- \n");
		clock_gettime(CLOCK_MONOTONIC, &ts_start_time);
		error = receiveFromCpld();
		clock_gettime(CLOCK_MONOTONIC, &ts_end_compute);
		total_fpga_computition_time = (ts_end_compute.tv_sec - ts_start_time.tv_sec)*1000000 + (ts_end_compute.tv_nsec - ts_start_time.tv_nsec) / 1000;
		printf("Total time required by fpga: 				%lf usec\n",total_fpga_computition_time);
		if(error)
		{
			printf("ERROR receiveFromFpga! \n");
			return (1);
		}

	}
}

